#include "hip/hip_runtime.h"
#!/usr/bin/env c-script


__global__
void
test(int *a,int *b){
  b[threadIdx.x]=a[threadIdx.x]*2;
} 

__global__
void
test2(int *a){
  a[threadIdx.x]*=2;
}

int
main(int argc,char** argv){
  //CUT_DEVICE_INIT(argc,argv);
  
  Mem<int> mem(12,"r");
  Mem<int> mem2(12);
  int a[1024];
  int b[1024];
  
  int i;
  for(i=0;i<mem.s;i++)
    mem[i]=i;

  for(i=0;i<sizeof(a)/sizeof(int);i++)
    a[i]=i;
  memcpy(b,a,sizeof(a));

  foreach(test,mem,mem2);
  foreach(test2,mem2);
  
  for(i=0;i<mem2.s;i++)
    printf("%d\n",mem2[i]);
  
  //  CUT_EXIT(argc,argv);
  return 0;
}

