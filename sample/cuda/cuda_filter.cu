#include "hip/hip_runtime.h"
#!/usr/bin/env c-script


#include <bmp.h>

struct img{
  int w;
  int h;
  int* d;
  __device__ __host__ img(){
    w=0;
    h=0;
    d=NULL;
  }
  
  __device__ __host__
  int&        operator () (int x,int y,int z){
    if(x>=w)
      x=w-1;
    else if(x<0)
      x=0;
    if(y>=h)
      y=h-1;
    else if(y<0)
      y=0;
    
    return d[(y*w+x)*3+z];
  }
  __device__ __host__
  const int&  operator () (int x,int y,int z)const{
    if(x>=w)
      x=w-1;
    else if(x<0)
      x=0;
    if(y>=h)
      y=h-1;
    else if(y<0)
      y=0;
    return d[(y*w+x)*3+z];
  }
};

void
copy(const BMP& bm,img& im){
  im.w=bm.w;
  im.h=bm.h;
  malloc_d(im.d,im.w*im.h*3*sizeof(int));
  cpy_h2d(bm.rgb,im.d,im.w*im.h*3*sizeof(int));
}
void
copy(const img& im,BMP& bm){
  bm.init(im.w,im.h);
  cpy_d2h(im.d,bm.rgb,im.w*im.h*3*sizeof(int));
}
void
copy(const img& im,img& im2){
  im2=im;
  malloc_d(im2.d,im.w*im.h*3*sizeof(int));
  cpy_d2d(im.d,im2.d,im.w*im.h*3*sizeof(int));
}


void
init(const img& im,img& im2){
  im2=im;
  malloc_d(im2.d,im.w*im.h*3*sizeof(int));
}

#define xl(in,x)  ((x)<0 ? 0 :( ((x)>=(in).w) ? (in).w-1:(x)))
#define yl(in,y)  ((y)<0 ? 0 :( ((y)>=(in).h) ? (in).h-1:(y)))
#define val(in,x,y,z) (in.d[((yl((in),(y)))*in.w+(xl((in),(x))))*3+(z)])

__global__
void
filter(img in,img out){
  int idx=blockIdx.x * blockDim.x + threadIdx.x;
  int x=idx%in.w;
  int y=idx/in.w;
  if(idx>in.w*in.h)
    return;
  for(int k=0;k<3;k++){
    int v=
      -in(x,y-1,k)
      -in(x-1,y,k)+in(x,y,k)*5-in(x+1,y,k)+
      -in(x,y+1,k);
    if(v>255)
      v=255;
    else if(v<0)
      v=0;
    out(x,y,k)=v;
  }
      
}


int
main(int argc,char** argv){
  BMP bin(argv[1]);
  BMP bout;

  img in;
  img out;

  copy(bin,in);
  init(in,out);

  
  int th=256;

  
  filter<<<(in.w*in.h+th-1)/th,th>>>(in,out);
  // filter(in,out);
  
  copy(out,bout);
  
  bout.write(argv[2]);

  free_d(in.d);
  free_d(out.d);
  
  return 0;
}

